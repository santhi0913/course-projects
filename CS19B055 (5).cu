#include<iostream>
#include<hip/hip_runtime.h>
#include <fstream>

using namespace std;


int **final;
class  Train
{
public:
    /* data */
    // int train_num;
    int num_class;
    int src;
    int dest;
    int *cls;

    void init_train(int num, int source, int destination){
        //  train_num = train;
         num_class = num;
         src = source;
         dest = destination;
         //cls = new int[num_class];
         cls = (int*)malloc(num_class*sizeof(int));
         
     }
   
};

class Request{
public:
    // int id;
    int train_num;
    int class_num;
    int source, desti, numSeats;
    void initialise( int tn , int cn, int src, int dest, int ns ){
        // id = id;
        train_num = tn;
        class_num = cn;
        source = src;
        desti = dest;
        numSeats = ns;
    }
};

// class Result
// {

// public:
//     int *occupied;
//     int *result;
//     void init_res(int num){

//     }
// };


__global__ void TrainTicket(int N, int R, Request *req,Train *train, int *result, int *occupied ){

    int tno = blockIdx.x*1000 + blockIdx.y;
    int cno = threadIdx.x;

    if(tno >= N) return;

    if(cno >= train[tno].num_class ) return;

    

    int start = train[tno].src;
    int base=(tno*25+cno)*51;
    

    for(int i=0; i<R; i++){
         
        if(req[i].train_num != tno || req[i].class_num != cno) continue;
        bool valid = true;

        for(int j= req[i].source- start ; j< req[i].desti-start; j++){
            if(occupied[base+j]+ req[i].numSeats > train[tno].cls[cno]){
                valid = false;
                result[i] =0;
                break;
            }
        }

        if(valid){
          for(int j= req[i].source- start ; j< req[i].desti-start; j++){
              occupied[base+j] += req[i].numSeats;
            }
            result[i] = 1;
        }

        
    }


}

// class Result{
//     public:
//     int *re;
//     int suc,fail;

//     void init_res(int n){
//         re = new int[n];
//     }
// };

void Major(int N){
    Train *train = new Train[N];
    Train *htrains = new Train[N];
    
    Train *d_train;
    int *d_occupied;    
    hipMalloc(&d_occupied, N*25*51*sizeof(int));
    hipMemset(d_occupied,0,N*25*51*sizeof(int)); 
    hipMalloc(&d_train, N* sizeof(Train));
    
    
    for(int i=0; i<N; i++){
        int tid,ncls,sr,ds;
        cin >> tid >> ncls >> sr >> ds;
        if(sr>ds) swap(sr,ds);
        train[tid].init_train(ncls,sr,ds);
       
        for(int j=0; j<ncls; j++){
            int cid, mc;
            cin >> cid >> mc;
            train[tid].cls[cid] = mc;

        }
    }

    memcpy(htrains,train,N*sizeof(Train));
    for(int i=0;i<N;i++) {
        hipMalloc(&htrains[i].cls ,train[i].num_class*sizeof(int));
        hipMemcpy(htrains[i].cls,train[i].cls,train[i].num_class*sizeof(int),hipMemcpyHostToDevice);
    }

    hipMemcpy(d_train,htrains,N*sizeof(Train),hipMemcpyHostToDevice);
    
    int* R;
    int B;
    Request **requuest;
    cin >> B;
    
    R = new int[B];
    requuest = new Request*[B];
    final = new int*[B];
    

    for(int i=0; i<B; i++){
        cin >> R[i];
        requuest[i] = new Request[R[i]];
        final[i] = new int[B];
        for(int j=0; j<R[i]; j++){
            int id,tn,cn,s,d,ns;
            cin >>id >>tn>>cn >> s >> d >> ns;
            if(s>d) swap(s,d);
            requuest[i][j].initialise(tn,cn,s,d,ns);
        }
    }

    

    for(int i=0; i<B; i++){
        Request* req = (Request*)malloc(R[i]*sizeof(Request));
        Request* d_req;
        hipMalloc(&d_req ,R[i]*sizeof(Request));
        int *res = (int*)malloc(R[i]*sizeof(int));
        int *d_result;
        hipMalloc(&d_result, R[i]*sizeof(int));
        hipMemcpy(d_result, res, R[i]*sizeof(int), hipMemcpyHostToDevice);

        for(int j=0; j<R[i];j++){
            req[j] = requuest[i][j];
        }

        hipMemcpy(d_req,req,R[i]*sizeof(Request),hipMemcpyHostToDevice);

        dim3 grid(ceil((double)N/(double)1000),1000,1);
        TrainTicket<<<grid,25>>>(N,R[i],d_req, d_train, d_result,d_occupied);
        hipDeviceSynchronize();

        hipMemcpy(res, d_result, R[i]*sizeof(int), hipMemcpyDeviceToHost);
        
        //for(int j=0; j<R[i]; j++) final[i][j] = res[j];
        int nums=0, numf=0,sum=0;
        for(int j=0; j<R[i]; j++ ){
            // cout << res[j]<< " ";
            if(res[j] == 1) {
                cout << "success\n";
                nums++;
                sum += (req[j].numSeats )* (abs(req[j].source - req[j].desti));
            }
            else{
                cout << "failure\n";
                numf++;
            }
        }
        cout << nums << " " << numf << endl;
        
        cout << sum << endl;
        
    }
}
   


int main(){
    

    int N;
    cin >> N;
    Major(N);
    // for(int i=0; i<B; i++){
    //     int nums=0, numf=0,sum=0;
    //     for(int j=0; j<R[i]; j++ ){
    //         // cout << res[j]<< " ";
    //         if(final[i][j] == 1) {
    //             cout << "success\n";
    //             nums++;
    //             sum += (requuest[i][j].numSeats )* (abs(requuest[i][j].source - requuest[i][j].desti));
    //         }
    //         else{
    //             cout << "failure\n";
    //             numf++;
    //         }
    //     }
    //     cout << nums << " " << numf << endl;
        
    //     cout << sum << endl;
    // }
    
    return 0;
}
